#include "hip/hip_runtime.h"

#include "GPU_Face_Detect.cuh"

#include "hip/hip_runtime.h"

#include "lock.h"

#define CONSTANT_MEM_SIZE 32
__constant__ GPUHaarStageClassifier stageClassifiers[CONSTANT_MEM_SIZE];

texture<int, 2, hipReadModeElementType> sumImageRef;
texture<float, 2, hipReadModeElementType> sqSumImageRef;

void allocateGPUCascade(GPUHaarCascade &h_gpuCascade, GPUHaarCascade &dev_gpuCascade)
{
	// copy generic parameters
	dev_gpuCascade.flags = h_gpuCascade.flags;
	dev_gpuCascade.numOfStages = h_gpuCascade.numOfStages;
	dev_gpuCascade.orig_window_size = h_gpuCascade.orig_window_size;
	dev_gpuCascade.real_window_size = h_gpuCascade.real_window_size;
	dev_gpuCascade.img_window_size = h_gpuCascade.img_window_size;
	dev_gpuCascade.scale = h_gpuCascade.scale;
	dev_gpuCascade.totalNumOfClassifiers = h_gpuCascade.totalNumOfClassifiers;

	// Allocate space for device classifiers and copy classifiers from host
	size_t GPU_Classifier_Size = h_gpuCascade.totalNumOfClassifiers * sizeof(GPUHaarClassifier);
	HANDLE_ERROR( hipMalloc( (void**)&dev_gpuCascade.haar_classifiers, GPU_Classifier_Size ) );
	HANDLE_ERROR( hipMemcpy(dev_gpuCascade.haar_classifiers, h_gpuCascade.haar_classifiers, GPU_Classifier_Size, hipMemcpyHostToDevice));

	HANDLE_ERROR( hipMalloc( (void**)&dev_gpuCascade.scaled_haar_classifiers, GPU_Classifier_Size ) );
	HANDLE_ERROR( hipMemcpy(dev_gpuCascade.scaled_haar_classifiers, h_gpuCascade.scaled_haar_classifiers, GPU_Classifier_Size, hipMemcpyHostToDevice));


	if(h_gpuCascade.numOfStages > CONSTANT_MEM_SIZE)
	{
		printf("ERROR: Number of stages is larger than the max size of constant memory alloted");
		system("pause");
		return;
	}

	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( stageClassifiers), h_gpuCascade.haar_stage_classifiers, sizeof(GPUHaarStageClassifier) * h_gpuCascade.numOfStages ) );
}

void allocateIntegralImagesGPU(CvMat * sumImage, CvMat *sqSumImage, hipArray *dev_sumArray, hipArray * dev_sqSumArray)
{
	//===========================================================
	// Allocate & reference texture memory for sum integral image
	//===========================================================

	// Create channel descripition for texture( 1 channel 32 bits, type signed int)
	hipChannelFormatDesc sum_channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned );

	// Allocate device memory for sum image texture
	HANDLE_ERROR( hipMallocArray(&dev_sumArray, &sum_channelDesc, sumImage->width, sumImage->height));

	// Copy image data from OpenCv to device memory
	HANDLE_ERROR(hipMemcpy2DToArray(dev_sumArray, 0, 0, sumImage->data.i, sumImage->step, sumImage->width * sizeof(int), sumImage->height, hipMemcpyHostToDevice));
	
	// Set parameters for CUDA texture reference
	sumImageRef.addressMode[0] = hipAddressModeWrap;
	sumImageRef.addressMode[1] = hipAddressModeWrap;
	sumImageRef.filterMode = hipFilterModePoint; //hipFilterModeLinear
	sumImageRef.normalized = false;

	// Bind texture reference to our allocated device memory	
	HANDLE_ERROR( hipBindTextureToArray(sumImageRef, dev_sumArray, sum_channelDesc));

	//==================================================================
	// Allocate & reference texture memory for square sum integral image
	//==================================================================

	// Create channel descripition for texture( 1 channel 64 bits, type float)
	hipChannelFormatDesc sqSum_channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
	// Allocate device memory for sum image texture
	HANDLE_ERROR( hipMallocArray(&dev_sqSumArray, &sqSum_channelDesc, sqSumImage->width, sqSumImage->height));

	// Copy image data from OpenCv to device memory
	HANDLE_ERROR(hipMemcpy2DToArray(dev_sqSumArray, 0, 0, sqSumImage->data.fl, sqSumImage->step, sqSumImage->width * sizeof(int), sqSumImage->height, hipMemcpyHostToDevice));
	
	// Set parameters for CUDA texture reference
	sqSumImageRef.addressMode[0] = hipAddressModeWrap;
	sqSumImageRef.addressMode[1] = hipAddressModeWrap;
	sqSumImageRef.filterMode = hipFilterModeLinear;
	sqSumImageRef.normalized = false;

	// Bind texture reference to our allocated device memory
	HANDLE_ERROR( hipBindTextureToArray(sqSumImageRef, dev_sqSumArray, sqSum_channelDesc));
}

void releaseTextures()
{
	hipUnbindTexture(sumImageRef);
	hipUnbindTexture(sqSumImageRef);
}

//===============================================================================
//
//
//===============================================================================

__device__ float calculateMean(GPURect rect)
{
	int A = tex2D(sumImageRef, rect.x, rect.y);
	int B = tex2D(sumImageRef, rect.x + rect.width, rect.y);
	int C = tex2D(sumImageRef, rect.x + rect.width, rect.y + rect.height);
	int D = tex2D(sumImageRef, rect.x, rect.y + rect.height);

	return (float)(A - B + C - D);
}


__device__ float calculateSum(GPURect rect, int win_start_x, int win_start_y)
{
	float tx = win_start_x + rect.x;
	float ty = win_start_y + rect.y;

	int A = tex2D(sumImageRef, tx, ty);
	int B = tex2D(sumImageRef, tx + rect.width, ty);
	int C = tex2D(sumImageRef, tx + rect.width, ty + rect.height);
	int D = tex2D(sumImageRef, tx, ty + rect.height);

	return (float)(A - B + C - D);
}

__device__ int getOffset(int x, int y)
{
	// blockDim.x * gridDim.x; = img.width
	return x + y * blockDim.x * gridDim.x;
}

__device__ float runHaarFeature(GPUHaarClassifier classifier, GPURect detectionWindow, float variance_norm_factor, float weightScale)
{
	double t = classifier.threshold * variance_norm_factor;

	double sum = calculateSum(classifier.haar_feature.rect0.r, detectionWindow.x, detectionWindow.y) * classifier.haar_feature.rect0.weight * weightScale;
	sum += calculateSum(classifier.haar_feature.rect1.r, detectionWindow.x, detectionWindow.y) * classifier.haar_feature.rect1.weight * weightScale;

	// If there is a third rect
	if(classifier.haar_feature.rect2.weight)
		sum += calculateSum(classifier.haar_feature.rect2.r, detectionWindow.x, detectionWindow.y) * classifier.haar_feature.rect2.weight * weightScale;
            
	if(sum >= t)
		return classifier.alpha1;
	else
		return classifier.alpha0;
}

__device__ float calculateVariance(GPURect detectionWindow)
{
	float inv_window_area = 1.0f / ((float)detectionWindow.width * detectionWindow.height);
	float weightScale = inv_window_area;

	// HaarCascade file requires normalization of features
	float mean = calculateMean(detectionWindow) * inv_window_area;
	
	float variance_norm_factor = tex2D(sqSumImageRef, detectionWindow.x, detectionWindow.y) - 
		tex2D(sqSumImageRef, detectionWindow.x + detectionWindow.width, detectionWindow.y) - 
		tex2D(sqSumImageRef, detectionWindow.x, detectionWindow.y + detectionWindow.height) + 
		tex2D(sqSumImageRef, detectionWindow.x + detectionWindow.width, detectionWindow.y + detectionWindow.height);
		
	variance_norm_factor = variance_norm_factor * inv_window_area - mean * mean;
	//variance_norm_factor = sqrt(variance_norm_factor * inv_window_area - mean * mean);

	if(variance_norm_factor >= 0.0f)
		variance_norm_factor = sqrt(variance_norm_factor);
	else
	{		
		variance_norm_factor = 1.0f;
	}

	return variance_norm_factor;
}



__global__ void haarDetection_v1(GPUHaarCascade haarCascade, GPURect * detectedFaces)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = getOffset(x,y);

	// If current pixel is out of bounds, just return
	
	if(x < haarCascade.img_detection_size.width || y < haarCascade.img_detection_size.height)
	{
		GPURect detectionWindow;
		detectionWindow.x = x;
		detectionWindow.y = y;
		detectionWindow.width = haarCascade.real_window_size.width;
		detectionWindow.height = haarCascade.real_window_size.height;

		float variance_norm_factor = calculateVariance(detectionWindow);
		float inv_window_area = 1.0f / ((float)detectionWindow.width * detectionWindow.height);
		float weightScale = inv_window_area;

		// Assume face was detected
		detectedFaces[offset] = detectionWindow;

		//bool faceDetected = true;
		// for each stage in cascade
		for(int i = 0; i < haarCascade.numOfStages; i++)
		{
			float stage_sum = 0.0;
			for(int j = 0; j < stageClassifiers[i].numofClassifiers; j++)
			{
				int index = j + stageClassifiers[i].classifierOffset;
				GPUHaarClassifier classifier = haarCascade.scaled_haar_classifiers[index];
				//GPUHaarClassifier classifier = haarCascade.haar_classifiers[index];

				stage_sum += runHaarFeature(classifier, detectionWindow, variance_norm_factor, inv_window_area);
			}

			// Classifier did not pass, abort entire cascade
			if( stage_sum < stageClassifiers[i].threshold)
			{
				// Set width to zero to indicate on CPU side that this is not a face
				detectedFaces[offset].width = 0;
				break;
			}
		}
	}
}

